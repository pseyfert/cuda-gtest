#include "hip/hip_runtime.h"
#include "cuda-gtest.h"

__global__ void DoNothing() {
}

TEST(TestWrapper, Pass) {
  ASSERT_LAUNCH_GOOD(DoNothing<<<1, 1>>>());
}

TEST(TestWrapper, Overcommit) {
  ASSERT_LAUNCH_BAD(DoNothing<<<4096, 4096>>>());
}
